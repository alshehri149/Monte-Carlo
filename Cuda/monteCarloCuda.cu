#include "hip/hip_runtime.h"
#include <math.h>
#include <nvtx3/nvToolsExt.h>
#include <hiprand.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include "../Common/option.h"

__global__ void reducef(float *array_in, double *reduct, size_t array_len)
{
	extern volatile __shared__ double sdata[];
	unsigned int blockSize = 128;
	size_t tid = threadIdx.x, gridSize = blockSize * gridDim.x, i = blockIdx.x * blockSize + tid;
	sdata[tid] = 0;
	while(i < array_len)
	{
		sdata[tid] += array_in[i];
		i += gridSize;
	}
	__syncthreads();
	if(blockSize >= 512) { if (tid < 256) sdata[tid] += sdata[tid + 256]; __syncthreads(); }
	if(blockSize >= 256) { if (tid < 128) sdata[tid] += sdata[tid + 128]; __syncthreads(); }
	if(blockSize >= 128) { if (tid <  64) sdata[tid] += sdata[tid + 64]; __syncthreads(); }
	
	if (tid < 32)
	{
		if(blockSize >= 64) sdata[tid] += sdata[tid + 32];
		if(blockSize >= 32) sdata[tid] += sdata[tid + 16];
		if(blockSize >= 16) sdata[tid] += sdata[tid + 8];
		if(blockSize >= 8) sdata[tid] += sdata[tid + 4];
		if(blockSize >= 4) sdata[tid] += sdata[tid + 2];
		if(blockSize >= 2) sdata[tid] += sdata[tid + 1];
	}
	if(tid == 0) reduct[blockIdx.x] = sdata[0];
}

__global__ void reduced(double *array_in, double *reduct, size_t array_len)
{
	extern volatile __shared__ double sdata[];
	unsigned int blockSize = 128;
	size_t tid = threadIdx.x, gridSize = blockSize * gridDim.x, i = blockIdx.x * blockSize + tid;
	sdata[tid] = 0;
	while(i < array_len)
	{
		sdata[tid] += array_in[i];
		i += gridSize;
	}
	__syncthreads();
	if(blockSize >= 512) { if (tid < 256) sdata[tid] += sdata[tid + 256]; __syncthreads(); }
	if(blockSize >= 256) { if (tid < 128) sdata[tid] += sdata[tid + 128]; __syncthreads(); }
	if(blockSize >= 128) { if (tid <  64) sdata[tid] += sdata[tid + 64]; __syncthreads(); }
	
	if (tid < 32)
	{
		if(blockSize >= 64) sdata[tid] += sdata[tid + 32];
		if(blockSize >= 32) sdata[tid] += sdata[tid + 16];
		if(blockSize >= 16) sdata[tid] += sdata[tid + 8];
		if(blockSize >= 8) sdata[tid] += sdata[tid + 4];
		if(blockSize >= 4) sdata[tid] += sdata[tid + 2];
		if(blockSize >= 2) sdata[tid] += sdata[tid + 1];
	}
	if(tid == 0) reduct[blockIdx.x] = sdata[0];
}

void mcs_build_rando_list(uint32_t num_sims, float* rando_list)
{
	hiprandGenerator_t randGenerator = {0};
	hiprandCreateGenerator(&randGenerator, HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandSetPseudoRandomGeneratorSeed(randGenerator, 0xdead5eed);
	hiprandGenerateNormal(randGenerator, rando_list, num_sims, 0, 1);
	hiprandDestroyGenerator(randGenerator);
}

//Calculate call and put prices with Monte Carlo method
__global__ void mcs_calc_price(uint32_t num_sims, option_t *opt, float *rando_list, float *call_vec, float *put_vec)
{
	/*Repeated exprerssions***************************************************/
	float S_adjust = opt->s * exp(opt->t * (opt->r - 0.5 * opt->v * opt->v));
	float xpr = sqrt(opt->v * opt->v * opt->t);
	/*************************************************************************/

	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if(index < num_sims)
	{
		float S_cur = S_adjust * exp(xpr * rando_list[index]);
		float call_max = S_cur - opt->k;
		float put_max = opt->k - S_cur;
		call_vec[index] = (call_max > 0) ? call_max : 0;
		put_vec[index] = (put_max > 0) ? put_max : 0;
	}
}

int main(int argc, char **argv)
{
	uint32_t SIM_COUNT = (argc > 1) ? atol(argv[1]) : 100000000;
	float *rando_list = NULL;
	float *call_vec = NULL;
	float *put_vec = NULL;
	double *call_red = NULL;
	double *put_red = NULL;
	option_t *option_d = NULL;
	option_t option = 
	{
		.s = 42,
		.k = 40,
		.r = 0.1,
		.v = 0.2,
		.t = 0.5,
		.call = 0,
		.put = 0
	};
	
	nvtxRangePush("TOTAL");
	//First we populate rando_list with randomly generated numbers
	nvtxRangePush("Rando_gen");
	hipMalloc((void**)&rando_list, sizeof(float) * SIM_COUNT);
	mcs_build_rando_list(SIM_COUNT, rando_list);
	//hipDeviceSynchronize();
	nvtxRangePop();//Rando_gen - pop
	
	//Then we calculate the call and put values
	nvtxRangePush("Price_calc");
	uint32_t threads = 128;
	uint32_t blocks = (SIM_COUNT + threads - 1) / threads;
	hipMalloc((void**)&call_vec, sizeof(float) * SIM_COUNT);
	hipMalloc((void**)&put_vec, sizeof(float) * SIM_COUNT);
	hipMalloc((void**)&option_d, sizeof(option_t));
	hipMemcpy(option_d, &option, sizeof(option_t), hipMemcpyHostToDevice);
	mcs_calc_price<<<blocks, threads>>>(SIM_COUNT, option_d, rando_list, call_vec, put_vec);
	
	//Reduce call_vec and put_vec and place result in call_red and put_red respectively
	hipMalloc((void**)&call_red, blocks * sizeof(double));
	hipMalloc((void**)&put_red, blocks * sizeof(double));
	//Reduce is called twice, the first time is to reduce each block of call_vec and put_vec in to a single block call_red and put_red respectively.
	//	The second is reduce the call_red and put_red to single values stored in (repurposed) call_vec and put_vec respectively.
	reducef<<< blocks, threads, threads * sizeof(double) >>> (call_vec, call_red, SIM_COUNT);
	reduced<<< 1, threads, threads * sizeof(double) >>> (call_red, (double*)call_vec, blocks);
	reducef<<< blocks, threads, threads * sizeof(double) >>> (put_vec, put_red, SIM_COUNT);
	reduced<<< 1, threads, threads * sizeof(double) >>> (put_red, (double*)put_vec, blocks);
	
	//Copy results back to host
	double temp1 = 0;
	double temp2 = 0;
	hipMemcpy(&option, option_d, sizeof(option_t), hipMemcpyDeviceToHost);
	hipMemcpy(&temp1, call_vec, sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(&temp2, put_vec, sizeof(double), hipMemcpyDeviceToHost);
	//Average the results
	option.call = (temp1 / (double)SIM_COUNT) * exp(-option.r * option.t);
	option.put = (temp2 / (double)SIM_COUNT) * exp(-option.r * option.t);
	nvtxRangePop();//Price_calc - pop
	nvtxRangePop();//TOTAL - pop

	//Finally we output the parameters and prices
	printf("Simulations count: %u\r\n", SIM_COUNT);
	printf("Spot price:        %f\r\n", option.s);
	printf("Strike price:      %f\r\n", option.k);
	printf("Risk-free rate:    %f\r\n", option.r);
	printf("Volatility:        %f\r\n", option.v);
	printf("Time to maturity:  %f\r\n", option.t);
	printf("\r\n");
	printf("\033[93;40mCall Price:        \033[92;40m%f\r\n", option.call);
	printf("\033[93;40mPut Price:         \033[91;40m%f\033[0m\r\n", option.put);
	
	hipFree(rando_list);
	hipFree(call_vec);
	hipFree(put_vec);
	hipFree(call_red);
	hipFree(put_red);
	hipFree(option_d);
	return 0;
}
